#include "../includes/headers.h"

int main(int argc, char** argv){
	
	const int NUM_VERTICES = 10000;
	const int VERTEX_BYTES = NUM_VERTICES * sizeof(int);
	const int NUM_EDGES = 10000;
	const int EDGE_BYTES = NUM_EDGES * sizeof(Edge);
	const int STARTING_VERTEX = 25;


	int h_vertices[NUM_VERTICES];

	
	for (int i = 0; i < NUM_VERTICES; ++i)
	{
		h_vertices[i] = i;
	}
		
	Edge h_edges[NUM_EDGES];
	
	
	for (int i = 0; i < NUM_VERTICES; ++i)
	{
		Edge* e = malloc(sizeof(Edge));
		e->first = (rand() % (NUM_VERTICES+1));
		e->first = (rand() % (NUM_VERTICES+1));
		memcpy(h_edges[i], e, sizeof(e));
	}	
	
	Edge* d_edges;
	int* d_vertices;

	hipMalloc((void**)&d_edges, EDGE_BYTES);
	hipMalloc((void**)&d_vertices, VERTEX_BYTES);

	hipMemcpy(d_edges, h_edges, EDGE_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_vertices, h_vertices, VERTEX_BYTES, hipMemcpyHostToDevice);

	initialize_vertices<<<10, NUM_VERTICES>>>(d_vertices, STARTING_VERTEX, NUM_VERTICES);
	
	
	bool* h_done;
	bool* d_done;
	int* d_current_depth;
	int* h_current_depth;

	&h_current_depth = 0;
	&h_done = true;
	
	while(!h_done){
		hipMemcpy(&d_done, &h_done, sizeof(bool), cudaHostToDevice);
		hipMemcpy(&d_current_depth, &h_current_depth, sizeof(int), cudaHostToDevice);
		bfs<<<10, NUM_EDGES>>>(h_edges, h_vertices, d_current_depth);
		hipMemcpy(&h_done, &d_done, sizeof(bool), cudaHostToDevice);
		hipMemcpy(&h_current_depth, &d_current_depth, sizeof(int), cudaHostToDevice);
	}

	
	hipFree(d_edges);
	hipFree(d_vertices);
	hipFree(d_done);
	
}