#include "../includes/headers.h"

int main(int argc, char** argv){
	
	const int NUM_VERTICES = 10240;
	const size_t VERTEX_BYTES = NUM_VERTICES * sizeof(int);
	const int NUM_EDGES = 10240;
	const size_t EDGE_BYTES = NUM_EDGES * sizeof(Edge);
	const int STARTING_VERTEX = 25;
	hipError_t err = hipSuccess;

	
	//declare the two arrays on host
	int h_vertices[NUM_VERTICES];
	Edge h_edges[NUM_EDGES];
	

	//fill up the edges array
	for (int i = 0; i < NUM_EDGES; ++i)   
	{
	    h_edges[i].first = (rand() % (NUM_VERTICES+1));
	    h_edges[i].second = (rand() % (NUM_VERTICES+1));
	}
	
	//define the two arrays on the device
	Edge* d_edges;
	int* d_vertices;

	//Allocate memory on device for both arrays
	err = hipMalloc((void**)&d_edges, EDGE_BYTES);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate edges array on device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMalloc((void**)&d_vertices, VERTEX_BYTES);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate vertices array on device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	

	err = hipMemcpy(d_edges, h_edges, EDGE_BYTES, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy edges array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMemcpy(d_vertices, h_vertices, VERTEX_BYTES, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vertices array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	//assign thread configuration
    int threadsPerBlock = 1024;
    int blocksPerGrid =(NUM_VERTICES + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		initialize_vertices<<<blocksPerGrid, threadsPerBlock>>>(d_vertices, STARTING_VERTEX);
	err = hipGetLastError();
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch initialization kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("Initialization completed\n");
	
	//Initialize depth counter
	int current_depth = 1;

	//Allocate and initialize done on host and device
	bool* d_done;
	bool h_done;
	err = hipMalloc((void**)&d_done, sizeof(bool));
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to allocte d_done(error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}
	

	for(;;){
		if(h_done == true) break;
		h_done = true;
		//printf("Entered while loop\n");
		err = hipMemcpy(d_done, &h_done, sizeof(bool), hipMemcpyHostToDevice);
		if (err != hipSuccess)
	    {
	        fprintf(stderr, "Failed to copy h_done to device(error code %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

	    printf("CUDA kernel launching with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		bfs<<<blocksPerGrid, threadsPerBlock>>>(d_edges, d_vertices, current_depth, d_done);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
	    {
	        fprintf(stderr, "Failed to launch bfs kernel (error code %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }
		//printf("Second kernel launch finished\n");

		err = hipMemcpy(&h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
	    {
	        fprintf(stderr, "Failed to copy d_done to host(error code %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

	    printf("BFS run for level %d\n", current_depth);
	    current_depth++;


	}
	//printf("Breadth first traversal completed over %d levels\n", h_current_depth);
	hipFree(d_edges);
	hipFree(d_vertices);
	//hipFree(d_done);
	//hipFree(d_current_depth);
	err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");

	return 0;
	
}