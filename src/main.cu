#include "../includes/headers.h"

int main(int argc, char** argv){
	
	const int NUM_VERTICES = 10000;
	const int VERTEX_BYTES = NUM_VERTICES * sizeof(int);
	const int NUM_EDGES = 10000;
	const int EDGE_BYTES = NUM_EDGES * sizeof(Edge);
	

	int h_vertices[NUM_VERTICES];

	
	Edge h_edges[NUM_EDGES];
	

	for (int i = 0; i < NUM_VERTICES; ++i)   
	{
	    h_edges[i].first = (rand() % (NUM_VERTICES+1));
	    h_edges[i].second = (rand() % (NUM_VERTICES+1));
	}
	
	Edge* d_edges;
	int* d_vertices;
	int* d_starting_vertex;
	int* h_starting_vertex;
	*h_starting_vertex = 25;

	hipMalloc((void**)&d_edges, EDGE_BYTES);
	hipMalloc((void**)&d_vertices, VERTEX_BYTES);

	hipMemcpy(d_edges, h_edges, EDGE_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_vertices, h_vertices, VERTEX_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(d_starting_vertex, h_starting_vertex, sizeof(int), hipMemcpyHostToDevice);

	initialize_vertices<<<10, NUM_VERTICES>>>(d_vertices, d_starting_vertex);
	
	
	bool* h_done;
	bool* d_done;
	int* d_current_depth;
	int* h_current_depth;

	hipMalloc((void**)&d_done, sizeof(bool));
	hipMalloc((void**)&d_current_depth, sizeof(int));

	*h_current_depth = 0;
	*h_done = true;
	
	while(!h_done){
		hipMemcpy(&d_done, &h_done, sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(&d_current_depth, &h_current_depth, sizeof(int), hipMemcpyHostToDevice);

		bfs<<<10, NUM_EDGES>>>(h_edges, h_vertices, d_current_depth, d_done);

		hipMemcpy(&h_done, &d_done, sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(&h_current_depth, &d_current_depth, sizeof(int), hipMemcpyHostToDevice);
	}

	hipFree(d_edges);
	hipFree(d_vertices);
	hipFree(d_done);
	hipFree(d_current_depth);
	
	
}