#include "hip/hip_runtime.h"
#include "../includes/headers.h"

int main(int argc, char** argv){
	
	//configurable parameters for data set
	const int NUM_VERTICES = 8192;
	const size_t VERTEX_BYTES = NUM_VERTICES * sizeof(int);
	const int NUM_EDGES = 524288;
	const size_t EDGE_BYTES = NUM_EDGES * sizeof(Edge);
	const int STARTING_VERTEX = 5571;
	hipError_t err = hipSuccess;
	
	//assign thread configuration
    int threadsPerBlock = 1024;
    int blocks =(NUM_VERTICES + threadsPerBlock - 1) / threadsPerBlock;
    //int edgeBlocks =(NUM_EDGES + threadsPerBlock - 1) / threadsPerBlock;
	clock_t begin, end;
	double time_spent;
	int edgeCounter= 0;
	
	//declare the two arrays on host
	int h_vertices[NUM_VERTICES];
	Edge h_edges[NUM_EDGES];
	
	//read file and write into host array
	FILE *infile;
    const char *path = "DataSet/8192-524288.txt";
    char line[100];
    int first, second;
    infile = fopen(path, "r");

  	if (!infile) {
    	printf("Couldn't open %s for reading\n", path);
    	exit(-1);
  	}
  
	while (fgets(line, sizeof(line), infile)!= NULL) 
	{
		
		sscanf(line, "%d\t%d", &first, &second);

	    h_edges[edgeCounter].first = first;
	    h_edges[edgeCounter].second = second;
	    
	    edgeCounter++;
	}
	
	fclose(infile);

	//debugging log to check that the array has been correctly written
	// for (int i = 0; i < NUM_EDGES; ++i)
	// {
	// 	printf("%d -> %d", h_edges[i].first, h_edges[i].second);
	// 	printf(((i % 4) != 3) ? "\t":"\n");
	// }
	
	
	//define pointers two device arrays
	Edge* d_edges;
	int* d_vertices;

	//allocate memory on device for both arrays
	err = hipMalloc((void**)&d_edges, EDGE_BYTES);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate edges array on device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMalloc((void**)&d_vertices, VERTEX_BYTES);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate vertices array on device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
   
    //copy vertices array from host to device
	err = hipMemcpy(d_vertices, h_vertices, VERTEX_BYTES, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vertices array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, threadsPerBlock);

		initialize_vertices<<<blocks, threadsPerBlock>>>(d_vertices, STARTING_VERTEX);

	err = hipGetLastError();
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch initialization kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	printf("Initialization completed\n");

	err = hipMemcpy(h_vertices, d_vertices, VERTEX_BYTES, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vertices array from device to kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //debugging log to check that the vertices has has been correctly initialized and copied back to host
	// for (int i = 0; i < NUM_VERTICES; ++i)
	// {
	// 	printf("%d : %d", i, h_vertices[i]);
	// 	printf(((i % 4) != 3) ? "\t":"\n");
	// }

    //copy host vertices and edges array to device and prepare to launch kernel
	err = hipMemcpy(d_vertices, h_vertices, VERTEX_BYTES, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vertices array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	err = hipMemcpy(d_edges, h_edges, EDGE_BYTES, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy edges array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	//Initialize depth counters
	int previous_depth = 0;
	int current_depth = 1;

	//Allocate and initialize termination variable modified on host and device
	int* d_modified;
	int h_modified;
	err = hipMalloc((void**)&d_modified, sizeof(int));
	if (err != hipSuccess)
	{
	    fprintf(stderr, "Failed to allocte d_done(error code %s)!\n", hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}
	
	begin = clock();

	do{
		
		h_modified = 0;
		
		err = hipMemcpy(d_modified, &h_modified, sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess)
	    {
	        fprintf(stderr, "Failed to copy h_done to device(error code %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

	    //printf("CUDA kernel launching with %d blocks of %d threads\n", vertexBlocks, threadsPerBlock);

		bfs<<<blocks, threadsPerBlock>>>(d_edges, d_vertices, previous_depth, current_depth, d_modified);
		hipDeviceSynchronize();

		err = hipGetLastError();
		if (err != hipSuccess)
	    {
	        fprintf(stderr, "Failed to launch bfs kernel (error code %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

		err = hipMemcpy(&h_modified, d_modified, sizeof(int), hipMemcpyDeviceToHost);
		
		if (err != hipSuccess)
	    {
	        fprintf(stderr, "Failed to copy d_done to host(error code %s)!\n", hipGetErrorString(err));
	        exit(EXIT_FAILURE);
	    }

	    //printf("BFS run for level %d\n", current_depth);


	    previous_depth++;
	    current_depth++;


	}while(h_modified != 0);
	
	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Time taken: %f\n", time_spent);
	hipFree(d_edges);
	hipFree(d_vertices);
	//hipFree(d_done);
	//hipFree(d_current_depth);
	err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");

	return 0;
	
}